
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int N=10;

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}


int main(){

int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );
    cudaCheckErrors("cudamalloc fail");

    for (int i=0; i<N; i++) {
        a[i] = -i; b[i] = i * i;
    }
 hipMemcpy ( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy ( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
    cudaCheckErrors("cuda memcpy fail");

    add<<<1,N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost );
    cudaCheckErrors("cudamemcpy or cuda kernel fail");
    for (int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i],b[i],c[i]);
    }

    hipFree (dev_a); hipFree (dev_b); hipFree (dev_c);

    return 0;

}
